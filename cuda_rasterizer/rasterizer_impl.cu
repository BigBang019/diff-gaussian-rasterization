#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "rasterizer_impl.h"
#include <iostream>
#include <fstream>
#include <algorithm>
#include <numeric>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#include "auxiliary.h"
#include "forward.h"
#include "backward.h"

// Helper function to find the next-highest bit of the MSB
// on the CPU.
uint32_t getHigherMsb(uint32_t n)
{
	uint32_t msb = sizeof(n) * 4;
	uint32_t step = msb;
	while (step > 1)
	{
		step /= 2;
		if (n >> msb)
			msb += step;
		else
			msb -= step;
	}
	if (n >> msb)
		msb++;
	return msb;
}

// Wrapper method to call auxiliary coarse frustum containment test.
// Mark all Gaussians that pass it.
__global__ void checkFrustum(int P,
	const float* orig_points,
	const float* viewmatrix,
	const float* projmatrix,
	bool* present)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	float3 p_view;
	present[idx] = in_frustum(idx, orig_points, viewmatrix, projmatrix, false, p_view);
}

// Generates one key/value pair for all Gaussian / tile overlaps. 
// Run once per Gaussian (1:N mapping).
/**
 * 构建映射：gaussian_keys_unsorted -> gaussian_values_unsorted
 * @param P                        [R] number of points
 * @param points_xy                [R] 点投影在image plane上的pixel index
 * @param depths                   [R] 点在camera coordinate下的z值，表示深度信息，可以用来表达点与点的前后关系
 * @param offsets                  [R] 影响矩形的面积的累加和
 * @param gaussian_keys_unsorted   [W] keys: 高32bit标记当前点影响哪个block，低32bit标记当前点的depth信息
 * @param gaussian_values_unsorted [W] values：标记当前点的idx
 * @param radii                    [R] 高斯投影在image plane影响的最大范围
 * @param grid                     [R] grid=((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1)
 */
__global__ void duplicateWithKeys(
	int P,
	const float2* points_xy,
	const float* depths,
	const uint32_t* offsets,
	uint64_t* gaussian_keys_unsorted,
	uint32_t* gaussian_values_unsorted,
	int* radii,
	dim3 grid)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Generate no key/value pair for invisible Gaussians
	if (radii[idx] > 0)
	{
		// Find this Gaussian's offset in buffer for writing keys/values.
		uint32_t off = (idx == 0) ? 0 : offsets[idx - 1];
		uint2 rect_min, rect_max;

		getRect(points_xy[idx], radii[idx], rect_min, rect_max, grid);

		// For each tile that the bounding rect overlaps, emit a 
		// key/value pair. The key is |  tile ID  |      depth      |,
		// and the value is the ID of the Gaussian. Sorting the values 
		// with this key yields Gaussian IDs in a list, such that they
		// are first sorted by tile and then by depth. 
		for (int y = rect_min.y; y < rect_max.y; y++)
		{
			for (int x = rect_min.x; x < rect_max.x; x++)
			{
				uint64_t key = y * grid.x + x;
				key <<= 32;
				key |= *((uint32_t*)&depths[idx]);
				gaussian_keys_unsorted[off] = key;
				gaussian_values_unsorted[off] = idx;
				off++;
			}
		}
	}
}

// Check keys to see if it is at the start/end of one tile's range in 
// the full sorted list. If yes, write start/end of this tile. 
// Run once per instanced (duplicated) Gaussian ID.
/**
 * 统计image plane的pixel被哪些射线影响。
 * point_list_keys记录射线keys
 * 统计结果的存在ranges里，ranges的idx意义是image plane的pixel坐标，value的意义是射线的idx范围[x,y)
 * @param L               [R] 点-block的影响关系总数
 * @param point_list_keys [R] ascending排序过后的keys
 * @param ranges          [W] idx为block id，记录当前block被point_list_keys的哪些射线影响：[x,y)
 */
__global__ void identifyTileRanges(int L, uint64_t* point_list_keys, uint2* ranges)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= L)
		return;

	// Read tile ID from key. Update start/end of tile range if at limit.
	uint64_t key = point_list_keys[idx];
	uint32_t currtile = key >> 32;
	if (idx == 0)
		ranges[currtile].x = 0;
	else
	{
		uint32_t prevtile = point_list_keys[idx - 1] >> 32;
		if (currtile != prevtile)
		{
			ranges[prevtile].y = idx;
			ranges[currtile].x = idx;
		}
	}
	if (idx == L - 1)
		ranges[currtile].y = L;
}

// Mark Gaussians as visible/invisible, based on view frustum testing
void CudaRasterizer::Rasterizer::markVisible(
	int P,
	float* means3D,
	float* viewmatrix,
	float* projmatrix,
	bool* present)
{
	checkFrustum << <(P + 255) / 256, 256 >> > (
		P,
		means3D,
		viewmatrix, projmatrix,
		present);
}

CudaRasterizer::GeometryState CudaRasterizer::GeometryState::fromChunk(char*& chunk, size_t P)
{
	GeometryState geom;
	obtain(chunk, geom.depths, P, 128);
	obtain(chunk, geom.clamped, P * 3, 128);
	obtain(chunk, geom.internal_radii, P, 128);
	obtain(chunk, geom.means2D, P, 128);
	obtain(chunk, geom.cov3D, P * 6, 128);
	obtain(chunk, geom.conic_opacity, P, 128);
	obtain(chunk, geom.rgb, P * 3, 128);
	obtain(chunk, geom.tiles_touched, P, 128);
	hipcub::DeviceScan::InclusiveSum(nullptr, geom.scan_size, geom.tiles_touched, geom.tiles_touched, P);
	obtain(chunk, geom.scanning_space, geom.scan_size, 128);
	obtain(chunk, geom.point_offsets, P, 128);
	return geom;
}

CudaRasterizer::ImageState CudaRasterizer::ImageState::fromChunk(char*& chunk, size_t N)
{
	ImageState img;
	obtain(chunk, img.accum_alpha, N, 128);
	obtain(chunk, img.n_contrib, N, 128);
	obtain(chunk, img.ranges, N, 128);
	return img;
}

CudaRasterizer::BinningState CudaRasterizer::BinningState::fromChunk(char*& chunk, size_t P)
{
	BinningState binning;
	obtain(chunk, binning.point_list, P, 128);
	obtain(chunk, binning.point_list_unsorted, P, 128);
	obtain(chunk, binning.point_list_keys, P, 128);
	obtain(chunk, binning.point_list_keys_unsorted, P, 128);
	hipcub::DeviceRadixSort::SortPairs(
		nullptr, binning.sorting_size,
		binning.point_list_keys_unsorted, binning.point_list_keys,
		binning.point_list_unsorted, binning.point_list, P);
	obtain(chunk, binning.list_sorting_space, binning.sorting_size, 128);
	return binning;
}

// Forward rendering procedure for differentiable rasterization
// of Gaussians.
/**
 *
 * @param geometryBuffer resize function of geometryBuffer
 * @param binningBuffer  resize function of binningBuffer
 * @param imageBuffer    resize function of imageBuffer
 * @param P              [R] number of points
 * @param D              [R] maximum spherical harmonic degree
 * @param M              [R] number of spherical coefficients
 * @param background     [R] (3, )
 * @param width          [R] image width
 * @param height         [R] image height
 * @param means3D        [R] (P, 3) original point set
 * @param shs            [R] (P, 1, 3) spherical harmonic coefficients
 * @param colors_precomp [R] (0) or ?
 * @param opacities      [R] (P, 1)
 * @param scales         [R] (P, 3)
 * @param scale_modifier
 * @param rotations      [R] (P, 4)
 * @param cov3D_precomp  [R] (0) or ?
 * @param viewmatrix     [R] (4, 4) 旋转矩阵，从world coordinate转化到camera coordinate
 * @param projmatrix     [R] (4, 4) 投影矩阵，从world coordinate转化到image plane上
 * @param cam_pos        [R] (3, )
 * @param tan_fovx
 * @param tan_fovy
 * @param prefiltered
 * @param out_color      (3, H, W)
 * @param out_depth      (2, H, W)
 * @param radii          (P, )
 * @return
 */
int CudaRasterizer::Rasterizer::forward(
	std::function<char* (size_t)> geometryBuffer,
	std::function<char* (size_t)> binningBuffer,
	std::function<char* (size_t)> imageBuffer,
	const int P, int D, int M,
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	const float* opacities,
	const float* scales,
	const float scale_modifier,
	const float* rotations,
	const float* cov3D_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* cam_pos,
	const float tan_fovx, float tan_fovy,
	const bool prefiltered,
	float* out_color,
	float* out_depth,
	int* radii)
{
	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	size_t chunk_size = required<GeometryState>(P);
	char* chunkptr = geometryBuffer(chunk_size);
	GeometryState geomState = GeometryState::fromChunk(chunkptr, P);

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Dynamically resize image-based auxiliary buffers during training
	int img_chunk_size = required<ImageState>(width * height);
	char* img_chunkptr = imageBuffer(img_chunk_size);
	ImageState imgState = ImageState::fromChunk(img_chunkptr, width * height);

	if (NUM_CHANNELS != 3 && colors_precomp == nullptr)
	{
		throw std::runtime_error("For non-RGB, provide precomputed Gaussian colors!");
	}

	// Run preprocessing per-Gaussian (transformation, bounding, conversion of SHs to RGB)
	FORWARD::preprocess(
		P, D, M,
		means3D,
		(glm::vec3*)scales,
		scale_modifier,
		(glm::vec4*)rotations,
		opacities,
		shs,
		geomState.clamped,
		cov3D_precomp,
		colors_precomp,
		viewmatrix, projmatrix,
		(glm::vec3*)cam_pos,
		width, height,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		radii,
		geomState.means2D,
		geomState.depths,
		geomState.cov3D,
		geomState.rgb,
		geomState.conic_opacity,
		tile_grid,
		geomState.tiles_touched,
		prefiltered
	);

	// Compute prefix sum over full list of touched tile counts by Gaussians
	// E.g., [2, 3, 0, 2, 1] -> [2, 5, 5, 7, 8]
	hipcub::DeviceScan::InclusiveSum(geomState.scanning_space, geomState.scan_size,
		geomState.tiles_touched, geomState.point_offsets, P);

	// Retrieve total number of Gaussian instances to launch and resize aux buffers
	int num_rendered;
	hipMemcpy(&num_rendered, geomState.point_offsets + P - 1, sizeof(int), hipMemcpyDeviceToHost);

	int binning_chunk_size = required<BinningState>(num_rendered);
	char* binning_chunkptr = binningBuffer(binning_chunk_size);
	BinningState binningState = BinningState::fromChunk(binning_chunkptr, num_rendered);

	// For each instance to be rendered, produce adequate [ tile | depth ] key 
	// and corresponding dublicated Gaussian indices to be sorted
	duplicateWithKeys << <(P + 255) / 256, 256 >> > (
		P,
		geomState.means2D,
		geomState.depths,
		geomState.point_offsets,
		binningState.point_list_keys_unsorted,
		binningState.point_list_unsorted,
		radii,
		tile_grid
		);

	int bit = getHigherMsb(tile_grid.x * tile_grid.y);

	// Sort complete list of (duplicated) Gaussian indices by keys
	hipcub::DeviceRadixSort::SortPairs(
		binningState.list_sorting_space,
		binningState.sorting_size,
		binningState.point_list_keys_unsorted, binningState.point_list_keys,
		binningState.point_list_unsorted, binningState.point_list,
		num_rendered, 0, 32 + bit);

	hipMemset(imgState.ranges, 0, tile_grid.x * tile_grid.y * sizeof(uint2));

	// Identify start and end of per-tile workloads in sorted list
	if (num_rendered > 0)
		identifyTileRanges << <(num_rendered + 255) / 256, 256 >> > (
			num_rendered,
			binningState.point_list_keys,
			imgState.ranges
			);

	// Let each tile blend its range of Gaussians independently in parallel
	const float* feature_ptr = colors_precomp != nullptr ? colors_precomp : geomState.rgb;
	FORWARD::render(
		tile_grid, block,
		imgState.ranges,
		binningState.point_list,
		width, height,
		geomState.means2D,
		feature_ptr,
		geomState.depths,
		geomState.conic_opacity,
		imgState.accum_alpha,
		imgState.n_contrib,
		background,
		out_color,
		out_depth);

	return num_rendered;
}

// Produce necessary gradients for optimization, corresponding
// to forward render pass
/**
 *
 * @param P              [R] number of points
 * @param D              [R] maximum spherical harmonic degree
 * @param M              [R] number of spherical coefficients
 * @param R              [R] num_rendered
 * @param background     [R]
 * @param width          [R] image width
 * @param height         [R] image height
 * @param means3D        [R] (P, 3) original point set
 * @param shs            [R] (P, 1, 3) spherical harmonic coefficients
 * @param colors_precomp [R]
 * @param scales         [R] (P, 3)
 * @param scale_modifier [R]
 * @param rotations      [R] (P, 4)
 * @param cov3D_precomp  [R]
 * @param viewmatrix     [R] (4, 4) 旋转矩阵，从world coordinate转化到camera coordinate
 * @param projmatrix     [R] (4, 4) 投影矩阵，从world coordinate转化到image plane上
 * @param campos         [R] (3, )
 * @param tan_fovx
 * @param tan_fovy
 * @param radii          [R] 高斯投影在image plane影响的最大半径
 * @param geom_buffer    [R]
 * @param binning_buffer [R]
 * @param img_buffer     [R]
 * @param dL_dpix        [W]
 * @param dL_depths      [W]
 * @param dL_dmean2D     [W] (P, 3)
 * @param dL_dconic      [W] (P, 2, 2)
 * @param dL_dopacity    [W] (P, 1)
 * @param dL_dcolor      [W] (P, 3)
 * @param dL_dmean3D     [W] (P, 3)
 * @param dL_dcov3D      [W] (P, 6)
 * @param dL_dsh         [W] (P, M, 3)
 * @param dL_dscale      [W] (P, 3)
 * @param dL_drot        [W] (P, 4)
 */
void CudaRasterizer::Rasterizer::backward(
	const int P, int D, int M, int R,
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	const float* scales,
	const float scale_modifier,
	const float* rotations,
	const float* cov3D_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* campos,
	const float tan_fovx, float tan_fovy,
	const int* radii,
	char* geom_buffer,
	char* binning_buffer,
	char* img_buffer,
	const float* dL_dpix,
	const float* dL_depths,
	float* dL_dmean2D,
	float* dL_dconic,
	float* dL_dopacity,
	float* dL_dcolor,
	float* dL_dmean3D,
	float* dL_dcov3D,
	float* dL_dsh,
	float* dL_dscale,
	float* dL_drot)
{
	GeometryState geomState = GeometryState::fromChunk(geom_buffer, P);
	BinningState binningState = BinningState::fromChunk(binning_buffer, R);
	ImageState imgState = ImageState::fromChunk(img_buffer, width * height);

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	const dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	const dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Compute loss gradients w.r.t. 2D mean position, conic matrix,
	// opacity and RGB of Gaussians from per-pixel loss gradients.
	// If we were given precomputed colors and not SHs, use them.
	const float* color_ptr = (colors_precomp != nullptr) ? colors_precomp : geomState.rgb;
	const float* depth_ptr = geomState.depths;
	BACKWARD::render(
		tile_grid,
		block,
		imgState.ranges,
		binningState.point_list,
		width, height,
		background,
		geomState.means2D,
		geomState.conic_opacity,
		color_ptr,
		depth_ptr,
		imgState.accum_alpha,
		imgState.n_contrib,
		dL_dpix,
		dL_depths,
		(float3*)dL_dmean2D,
		(float4*)dL_dconic,
		dL_dopacity,
		dL_dcolor);

	// Take care of the rest of preprocessing. Was the precomputed covariance
	// given to us or a scales/rot pair? If precomputed, pass that. If not,
	// use the one we computed ourselves.
	const float* cov3D_ptr = (cov3D_precomp != nullptr) ? cov3D_precomp : geomState.cov3D;
	BACKWARD::preprocess(P, D, M,
		(float3*)means3D,
		radii,
		shs,
		geomState.clamped,
		(glm::vec3*)scales,
		(glm::vec4*)rotations,
		scale_modifier,
		cov3D_ptr,
		viewmatrix,
		projmatrix,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		(glm::vec3*)campos,
		(float3*)dL_dmean2D,
		dL_dconic,
		(glm::vec3*)dL_dmean3D,
		dL_dcolor,
		dL_dcov3D,
		dL_dsh,
		(glm::vec3*)dL_dscale,
		(glm::vec4*)dL_drot);
}
